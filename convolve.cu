#include "hip/hip_runtime.h"

/*
* Week 3
* Parallel Programming
* 2011-2012
* University of Birmingham
*
* This is a first step towards implementing "parallel reduce".
* Reducing means using an operation to aggregate the values of
* a data type, such an array or a list.
*
* For example, to calculate the sum we aggregate addition:
*     a1 + a2 + a3 + a4 ...
* To calculate the maximum we aggregate the max operation:
*     max (a1, max(a2, max(a3, ...
* Note that the order in which the device map, which is parallel,
* and the host map, which is sequential, will differ, therefore the
* operation needs to be associative.
* Operations such as +, * or max are associative, but function of
* two arguments, in general, are not!
*/


#include "cutil_inline.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>

using namespace std;

const int ITERS = 100;

/*
* Calculate the reduce by f of all elements in data_in and
* store the result at a location of your choice in in data_out.
*
* The initial implementation is correct but totally sequential,
* and it only uses one thread.
* Improve it, to take advantage of GPU parallelism.
* To ensure performance, identify and avoid divergences!
*
* THIS YOU NEED TO CHANGE!
*/
__global__ void convolve(float* data_in, float initial)
{
    //int tx = threadIdx.x;
    //int bk = blockIdx.x;
}

/*
* Reference CPU implementation, taken from http://www.songho.ca/dsp/convolution/convolution.html
*/
bool convolve1D(float* in, float* out, int dataSize, float* kernel, int kernelSize)
{
    int i, j, k;

    // check validity of params
    if(!in || !out || !kernel) return false;
    if(dataSize <=0 || kernelSize <= 0) return false;

    // start convolution from out[kernelSize-1] to out[dataSize-1] (last)
    for(i = kernelSize-1; i < dataSize; ++i)
    {
        out[i] = 0;                             // init to 0 before accumulate

        for(j = i, k = 0; k < kernelSize; --j, ++k)
        {
            out[i] += in[j] * kernel[k];
        }
    }

    // convolution from out[0] to out[kernelSize-2]
    for(i = 0; i < kernelSize - 1; ++i)
    {
        out[i] = 0;                             // init to 0 before sum

        for(j = i, k = 0; j >= 0; --j, ++k)
        {
            out[i] += in[j] * kernel[k];
        }
    }

    return true;
}

float *splitFloats(string line){
    std::vector<float> floats;
    int seen_whitespace = 1;
    for(std::string::size_type i = 0; i < line.size(); ++i) {
        if(line[i] == ' ') seen_whitespace = 1;
        if(seen_whitespace){
            floats.push_back(strtof(&line[i], NULL));
            seen_whitespace = 0;
        }

    }

    return &floats[0];
}

/*
* Main program and benchmarking
*/
int main(int argc, char** argv)
{
    int devID;
    hipDeviceProp_t props;

    // get number of SMs on this GPU
    cutilSafeCall(hipGetDevice(&devID));
    cutilSafeCall(hipGetDeviceProperties(&props, devID));

    string line;
    ifstream sample ("sample.txt");
    getline (sample,line);
    sample.close();

    // allocate host memory
    float *in = splitFloats(line);
    float out[5];
    float k[2] = {2,1};

    unsigned int timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

    convolve1D(in, out, 5, k, 2);

    for(int i = 0; i < 5; i++)
    {
        printf("%f, ", out[i]);
    }

    printf("\n");

    cutilCheckError(cutStopTimer(timer));
    //printf("%d \n", success);

    // allocate device memory
    //float* d_data_in;
    //cutilSafeCall(hipMalloc((void**) &d_data_in, mem_size));

    // copy host memory to device


    // set up kernel for execution
    //printf("Run %d Kernels.\n\n", ITERS);
    //unsigned int timer = 0;
    //cutilCheckError(cutCreateTimer(&timer));
    //cutilCheckError(cutStartTimer(timer));

    // execute kernel
    //for (int j = 0; j < ITERS; j++)
    //{
    //  cutilSafeCall(hipMemcpy(d_data_in, h_data_in,
    //  mem_size, hipMemcpyHostToDevice));

    //   reduce<<<GRID_SIZE, BLOCK_SIZE >>>(d_data_in, 0.0);
    //   reduce<<<GRID_SIZE, BLOCK_SIZE / 2>>>(d_data_in, 0.0);
    //   reduce<<<GRID_SIZE, BLOCK_SIZE / 4>>>(d_data_in, 0.0);


    // copy result from device to host
    //   cutilSafeCall(hipMemcpy(h_data_out, d_data_in,
    //   mem_size, hipMemcpyDeviceToHost));

    // Finish the reduction on the host to avoid the overhead of setting up the kernal for small n
    //h_data_out[0] = host_reduce(h_data_out, 0.0, VECTOR_SIZE / 8);

    //  }

    // check if kernel execution generated and error
    //  cutilCheckMsg("Kernel execution failed");

    // wait for device to finish
    //  hipDeviceSynchronize();

    // stop and destroy timer
    //  cutilCheckError(cutStopTimer(timer));
    //  double dSeconds = cutGetTimerValue(timer)/(1000.0);
    //  double dNumOps = ITERS * size;
    //  double gflops = dNumOps/dSeconds/1.0e9;

    //Log througput
    //  printf("Throughput = %.4f GFlop/s\n", gflops);
    //  cutilCheckError(cutDeleteTimer(timer));



    // error check
    //  printf("Host reduce   : %.4f\n", host_reduce(h_data_in, 0.0, VECTOR_SIZE));
    //  printf("Device reduce : %.4f\n", h_data_out[0]);

    // clean up memory
    //  free(h_data_in);
    //  free(h_data_out);
    //  cutilSafeCall(hipFree(d_data_in));

    // exit and clean up device status
    //  hipDeviceReset();
}
